#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"


//declare matrix
void print(double *C, int size );
int main(int argc, char const *argv[])
{
    //Declare number of streams
    int num_streams, N, size;

    //get values
    N = atoi(argv[1]);
    num_streams = atoi(argv[2]);

    //calculate size in bytes
    size = N * N * sizeof(double);

    // cuda create event 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    //allocate CPU memory
    double *A, *B *C;
    
    hipHostAlloc((void **)&A, size,hipHostMallocDefault);
    hipHostAlloc((void **)&B, size,hipHostMallocDefault);
    hipHostAlloc((void **)&C, size,hipHostMallocDefault);
    
    //Allocate GPU memory
    double *d_A,*d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    //initialize A,B,C 
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
          A[i + j*N] = 0.5;
          B[i + j*N] = 0.25;
          C[i + j* N] = 0.0;
        }
    }
    // set alphars
    const double alpha = 1.0;
    const double beta = 0.0;
    
    //create handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    //Create streams 
    hipStream_t streams[num_streams];
    
    //compute
    hipEventRecord(start, 0);
    for (int i = 0; i < num_streams; i++) 
    {
        //printf("%g \n",0.0);
        hipStreamCreate(&streams[i]);
        hipMemcpyAsync(d_A , A ,size, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_B , B ,size, hipMemcpyHostToDevice, streams[i]);
       // hipMemcpyAsync(d_C , C ,size, hipMemcpyHostToDevice, streams[i]);
        hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,N,N,N,&alpha,d_A,N,d_B,N,&beta,d_C,N);
        hipMemcpyAsync(C, d_C ,size, hipMemcpyDeviceToHost, streams[i]);
        hipStreamSynchronize (streams[i]);
       
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    //Destroy handle
    hipblasDestroy(handle);

    //destory event
    hipEventDestroy(stop);
    hipEventDestroy(start);


    // destroy stream
    for (int i = 0; i < num_streams; ++i)
    {
        hipStreamDestroy(streams[i]);
    }

    // print elapsed time
    printf("%d %d% .6f %g \n",N, num_streams, elapsedTime / 1000.0f, num_streams * 2e-9 * N*N*N /(elapsedTime / 1000.0f));
    //print(C,5);

    //Free CPU memory 
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);

    //Freee GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}

// print results of the array
void print(double *C, int size )
{
 int i,j;
 for(i=0; i < size; i++)
  {
    for(j=0; j < size; j++)
      printf("  %d", C[size*i+j]);
    printf("\n");
  }
}