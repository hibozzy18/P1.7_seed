#include <stdio.h>

#include <hip/hip_runtime.h>

__global__ void
add2(double *c, const double *a, const double *b) {
  c[0] = a[0] + b[0];
}

int
main(void) {
  double *a, *b, *c;
  float t;

  hipEvent_t start, stop;

  hipMalloc( (void **)&a, sizeof(double) );
  hipMalloc( (void **)&b, sizeof(double) ) ;
  hipMalloc( (void **)&c, sizeof(double) ) ;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  add2<<<1,1>>>(c, a, b);
  hipEventRecord(stop, 0);

  hipEventSynchronize(stop);

  hipEventElapsedTime(&t, start, stop);
  printf("cudaEventElapsedTime(%.6f)\n", t / 1000.0f);

  hipFree(c);
  hipFree(b);
  hipFree(a);

  return 0;
}
