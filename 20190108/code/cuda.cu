#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>

// print to check printed array 

printArray(double * Array);

//Implement the matmul method on GPU
__global__ void matmul(double *d_A, double *d_B, double *d_E, int size)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < size)
    E[idx] = A[idx] * B[idx];
}

// implement the AddArray on GPU
__global__ void Addarray(double *d_E, double *d_C, double *d_D, int size)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < size);
    D[idx] = C[idx] + E[idx];
}

int main(int argc, char *argv[])
{
    //declare cuda variables
    double *d_A, *d_B, *d_E, *d_C, *d_D;

    //declare CPU variables
    double A, B, E, C, D;

    //Declare variables 
    int size_bytes, NUM_BLOCKS, NUM_THREADS, N;
    size_bytes = size * sizeof(double);
    
    //initialize
    N = atoi(argv[1]);
    const int NUM_THREADS = 32;
    const int NUM_BLOCKS = ( N + (N-1) ) / NUM_THREADS;

    //create Streams
    hipStream_t stream1, stream2;
    hipStreamCreate( &stream1);
    hipStreamCreate( &stream2);

    //create Events
    hipEvent_t e1,e2,e3;
    hipEventCreateWithFlags(&e1, hipEventDisableTiming);
    hipEventCreateWithFlags(&e2, hipEventDisableTiming);
    hipEventCreateWithFlags(&e3, hipEventDisableTiming);
    
    //Allocate GPU memory 
    hipMalloc((void**)&d_A, size_bytes);
    hipMalloc((void**)&d_B, size_bytes);
    hipMalloc((void**)&d_C, size_bytes);
    hipMalloc((void**)&d_D, size_bytes);
    hipMalloc((void**)&d_E, size_bytes);

    //Allocate host memory
    hipHostAlloc((void **)&A, size_bytes),hipHostMallocDefault);
    hipHostAlloc((void **)&B, size_bytes),hipHostMallocDefault);
    hipHostAlloc((void **)&C, size_bytes),hipHostMallocDefault);
    hipHostAlloc((void **)&D, size_bytes),hipHostMallocDefault);
    hipHostAlloc((void **)&E, size_bytes),hipHostMallocDefault);

    //initialize A,B,C 
    for(int i=0; i<size; i++)
    {
        A[i] = 1.0;
        B[i] = 1.0;
        C[i] = 1.0;
    }

    //load A and B from CPU to GPU S1
    hipMemcpyAsync(d_A,A,size_bytes,hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_B,B,size_bytes,hipMemcpyHostToDevice, stream1);
    hipEventRecord(e1,stream1);

    // Multiply S1
    matmul<<<NUM_BLOCKS, NUM_THREADS, 0, stream1>>>(d_A,d_B,d_E);
    hipEventRecord(e2,stream1);

    //load C from CPU to GPU using  S2
    hipStreamWaitEvent( stream2 , e1 , 0);
    hipMemcpyAsync(d_C,C,size_bytes,hipMemcpyHostToDevice, stream2);

    //Add E to C S2
    hipStreamWaitEvent( stream2 , e2 , 0);
    Addarray<<<NUM_BLOCKS, NUM_THREADS,0, stream2>>>(d_E,d_C,d_D);

    //Load E from GPU to CPU S1
    hipMemcpyAsync(E,d_E,size_bytes,hipMemcpyDeviceToHost, stream1);

    //print the array to check the correctness
    printArray(E);

    //load D to CPU from GPU using S2
    hipMemcpyAsync(D,d_D,size_bytes,hipMemcpyDeviceToHost, stream2);

    //print the array to check the correctness  
    printArray(D);

    //free memory from GPU and CPU
    hipHostFree(A); hipFree(d_A);
    hipHostFree(B); hipFree(d_B);
    hipHostFree(C); hipFree(d_C);
    hipHostFree(D); hipFree(d_D);
    hipHostFree(E); hipFree(d_E);

    //destroy Stream
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    //destroy Events
    hipEventDestroy(e1);
    hipEventDestroy(e2);
    hipEventDestroy(e3);

    return 0
}

//verify correctness of multiplication
printArray(double * Array)
{
    for(int i =0; i<size; i++)
    {
        printf("%d", Array[i])
    }
}
