#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>

# define NUM_THREADS 16 
# define NUM_BLOCKS  8

// print to check printed array 

printArray(double * Array);

__global__ void matmul(double *d_A, double *d_B, double *d_E)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < size)
    E[idx] = A[idx] * B[idx];
}

__global__ void Addarray(double *d_E, double *d_C, double *d_D)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < size)
    D[idx] = C[idx] + E[idx];
}


int main(int argc, char *argv[])
{
    //declare cuda variables
    double *d_A, *d_B, *d_E, *d_C, *d_D;

    //declare CPU variables
    double A, B, E, C, D;

    //
    hipDeviceProp_t prop;
    int dev_id;
    hipGetDevice( &dev_id );
    hipGetDeviceProperties( &prop, dev_id );
    if (0 == prop.deviceOverlap) 
    {
         fprintf(stderr, "No handle overlap support");
    }

    //create Streams
    hipStream_t stream1, stream2;
    hipStreamCreate( &stream1);
    hipStreamCreate( &stream2);

    //create Events
    hipEvent_t e1,e2,e3;
    hipEventCreateWithFlags(&e1, hipEventDisableTiming);
    hipEventCreateWithFlags(&e2, hipEventDisableTiming);
    hipEventCreateWithFlags(&e3, hipEventDisableTiming);
    

    //Allocate GPU memory 
    hipMalloc((void**)&d_A, size * sizeof(double));
    hipMalloc((void**)&d_B, size * sizeof(double));
    hipMalloc((void**)&d_C, size * sizeof(double));
    hipMalloc((void**)&d_D, size * sizeof(double));
    hipMalloc((void**)&d_E, size * sizeof(double));

    //Allocate host memory
    hipHostAlloc((void **)&A, size * sizeof(double)),hipHostMallocDefault);
    hipHostAlloc((void **)&B, size * sizeof(double)),hipHostMallocDefault);
    hipHostAlloc((void **)&C, size * sizeof(double)),hipHostMallocDefault);
    hipHostAlloc((void **)&D, size * sizeof(double)),hipHostMallocDefault);
    hipHostAlloc((void **)&E, size * sizeof(double)),hipHostMallocDefault);

    //initialize A,B,C 
    for(int i=0; i<size; i++)
    {
        A[i] = 1.0;
        B[i] = 1.0;
        C[i] = 1.0;
    }

    //load A and B from CPU to GPU S1
    hipMemcpyAsync(d_A,A,size*sizeof(double),hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_B,B,size*sizeof(double),hipMemcpyHostToDevice, stream1);
    hipEventRecord(e1,stream1);

    // Multiply S1
    matmul<<<NUM_BLOCKS, NUM_THREADS, 0, stream1>>>(d_A,d_B,d_E);
    hipEventRecord(e2,stream1);

    //load C from CPU to GPU using  S2
    hipStreamWaitEvent( stream2 , e1 , 0);
    hipMemcpyAsync(d_C,C,size*sizeof(double),hipMemcpyHostToDevice, stream2);

    //Add E to C S2
    hipStreamWaitEvent( stream2 , e2 , 0);
    Addarray<<<NUM_BLOCKS, NUM_THREADS,0, stream2>>>(d_E,d_C,d_D);

    //Load E from GPU to CPU S1
    hipMemcpyAsync(E,d_E,size*sizeof(double),hipMemcpyDeviceToHost, stream1);

    //print the array to check the correctness
    printArray(E);

    //load D to CPU from GPU using S2
    hipMemcpyAsync(D,d_D,size*sizeof(double),hipMemcpyDeviceToHost, stream2);

    //print the array to check the correctness  
    printArray(D);

    //free memory from GPU and CPU
    hipHostFree(A); hipFree(d_A);
    hipHostFree(B); hipFree(d_B);
    hipHostFree(C); hipFree(d_C);
    hipHostFree(D); hipFree(d_D);
    hipHostFree(E); hipFree(d_E);

    //destroy Stream
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    //destroy Events
    hipEventDestroy(e1);
    hipEventDestroy(e2);
    hipEventDestroy(e3);

    return 0
}

//verify correctness of multiplication
printArray(double * Array)
{
    for(int i =0; i<size; i++)
    {
        printf("%d", Array[i])
    }
}
