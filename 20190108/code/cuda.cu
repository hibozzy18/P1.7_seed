#include<stdio.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

// print to check  array 
void check(int dim, double* A, double* B, double* C, double* D, double* E );
void check_mult(int size, double* A, double* B, double* E );

//Implement the matmul method on GPU
__global__ void matmul(double *d_A, double *d_B, double *d_E, int size)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < size)
    d_E[idx] = d_A[idx] * d_B[idx];
}

// implement the AddArray on GPU
__global__ void Addarray(double *d_E, double *d_C, double *d_D, int size)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < size);
    d_D[idx] = d_C[idx] + d_E[idx];
}

int main(int argc, char *argv[])
{
    //declare cuda variables
    double *d_A, *d_B, *d_E, *d_C, *d_D;

    //declare CPU variables
    double *A, *B, *E, *C, *D;

    //Declare variables 
    int size_bytes, N;
    
    //initialize
    N = atoi(argv[1]);

    size_bytes = N* N * sizeof(double);
    const int NUM_THREADS = 32;
    const int NUM_BLOCKS = ( N + (NUM_THREADS-1) ) / NUM_THREADS;

    //create Streams
    hipStream_t stream1, stream2;
    hipStreamCreate( &stream1);
    hipStreamCreate( &stream2);

    //create Events
    hipEvent_t e1,e2;
    hipEventCreateWithFlags(&e1, hipEventDisableTiming);
    hipEventCreateWithFlags(&e2, hipEventDisableTiming);
    //cudaEventCreateWithFlags(&e3, cudaEventDisableTiming);
    
    //Allocate GPU memory 
    hipMalloc((void **)&d_A, size_bytes);
    hipMalloc((void **)&d_B, size_bytes);
    hipMalloc((void **)&d_C, size_bytes);
    hipMalloc((void **)&d_D, size_bytes);
    hipMalloc((void **)&d_E, size_bytes);

    //Allocate host memory
    hipHostAlloc((void **)&A, size_bytes,hipHostMallocDefault);
    hipHostAlloc((void **)&B, size_bytes,hipHostMallocDefault);
    hipHostAlloc((void **)&C, size_bytes,hipHostMallocDefault);
    hipHostAlloc((void **)&D, size_bytes,hipHostMallocDefault);
    hipHostAlloc((void **)&E, size_bytes,hipHostMallocDefault);

    //initialize A,B,C 
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
          A[i + j*N] = 0.5;
          B[i + j*N] = 0.25;
          C[i + j*N] = 0.0;
        }
    }

    //load A and B from CPU to GPU S1
    hipMemcpyAsync(d_A,A,size_bytes,hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(d_B,B,size_bytes,hipMemcpyHostToDevice, stream1);
    hipEventRecord(e1,stream1);

    // Multiply  using S1
    matmul<<<NUM_BLOCKS, NUM_THREADS, 0, stream1>>>(d_A,d_B,d_E,N);
    hipEventRecord(e2,stream1);

    //load C from CPU to GPU using  S2
    hipStreamWaitEvent( stream2 , e1 , 0);
    hipMemcpyAsync(d_C,C,size_bytes,hipMemcpyHostToDevice, stream2);

    //Add E to C S2
    hipStreamWaitEvent( stream2 , e2 , 0);
    Addarray<<<NUM_BLOCKS, NUM_THREADS,0, stream2>>>(d_E,d_C,d_D, N);

    //Load E from GPU to CPU S1
    hipMemcpyAsync(E,d_E,size_bytes,hipMemcpyDeviceToHost, stream1);

    //print the array to check the correctness
    check_mult(N, A, B, E );

    //load D to CPU from GPU using S2
    hipMemcpyAsync(D,d_D,size_bytes,hipMemcpyDeviceToHost, stream2);

    //print the array to check the correctness  
   // check( N, A, B, C, D, E );

    //free memory from GPU and CPU
    hipHostFree(A); hipFree(d_A);
    hipHostFree(B); hipFree(d_B);
    hipHostFree(C); hipFree(d_C);
    hipHostFree(D); hipFree(d_D);
    hipHostFree(E); hipFree(d_E);

    //destroy Stream
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    //destroy Events
    hipEventDestroy(e1);
    hipEventDestroy(e2);

    return 0;
}

//verify correctness of multiplication
void check_mult(int size, double* A, double* B, double* E )
{
	int i, e;
	for (i = 0; i < size; ++i)
	{
		e = A[i]*B[i];
        if(E[i] != e) 
        {
            printf("Wrong results \n"); 
            return;
        }
    }
    printf("Results are correct \n");
	return;
}


void check(int size, double* A, double* B, double* C, double* D, double* E )
{
	int i, e, d;
	for (i = 0; i < size; ++i)
	{
		e = A[i]*B[i];
		d = e + C[i];
        if(E[i] != e) 
        {
            printf("Wrong results \n"); 
            return;
        }
        if(D[i] != d) 
        {
            printf("wrong results \n"); 
            return;
        }
    }
    printf("Results are correct \n");
	return;
}